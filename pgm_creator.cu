#include "pgm.h"
#include <stdio.h>
#include <stdlib.h>

int main(int argc, char **argv) {
  if (argc != 4) {
    printf("Wrong usage: ./pgm_creator.out <width> <height>"
           "<output_filename>\n");
    return 0;
  }

  int width = atoi(argv[1]);
  int height = atoi(argv[2]);

  pgm_image image;

  create_random_pgm_image(&image, width, height);
  int32_t err = save_pgm_to_file(argv[3], &image);

  if (err != NO_ERR) {
    printf("ERR = %d\n", err);
    return 1;
  }

  return 0;
}
