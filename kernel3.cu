#include "hip/hip_runtime.h"
#include "kernels.h"

#define threads_block 512
#define thread_pixel 16 // Each thread works on 16 pixels
#include <stdio.h>

void run_kernel3(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
  // Calculate threads and blocks for processing pixels step.
  int num_threads, num_blocks;
  num_threads = (width * height + thread_pixel - 1) / thread_pixel;
  if (num_threads > threads_block) {
    num_blocks = (num_threads + threads_block - 1) / threads_block;
    num_threads = threads_block;
  }else {
    num_blocks = 1;
  }
  dim3 dimBlock(num_threads, 1, 1);
  dim3 dimGrid(num_blocks, 1, 1);
  kernel3<<<dimGrid, dimBlock>>>(filter, dimension, input, output, width, height);
  // Initialiaze golbal device variables to store minimum and maximum.
  int32_t *gmin = NULL;
  int32_t *gmax = NULL;
  hipMalloc((void **)&gmin, width * height * sizeof(int32_t));
  hipMalloc((void **)&gmax, width * height * sizeof(int32_t));
  hipMemcpy(gmin, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);
  hipMemcpy(gmax, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);

  // Calculate threads and blocks for reduction step.
  if (width * height > threads_block) {
    num_threads = threads_block;
    num_blocks = (width * height + threads_block - 1) / threads_block;
  }else {
    num_threads = width * height;
    num_blocks = 1;
  }
  dim3 newblock(num_threads, 1, 1);
  dim3 newgrid(num_blocks, 1, 1);
  int shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
  reduction<<<newgrid, newblock, shMemSize>>>(gmin, gmax, width * height);

  // Repeat calling reduction kernel to get final maximum and minimum.
  while (num_blocks != 1) {
    int n = num_blocks;
    if (num_blocks > threads_block) {
      num_threads = threads_block;
      num_blocks = (num_blocks + threads_block - 1) / threads_block;
    }else {
      num_threads = num_blocks;
      num_blocks = 1;
    }
    shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
    dim3 newblock(num_threads, 1, 1);
    dim3 newgrid(num_blocks, 1, 1);
    reduction<<<newgrid, newblock, shMemSize>>>(gmin, gmax, n);
  }
  // Calling normalize kernel.
  normalize3<<<dimGrid, dimBlock>>>(output, width, height, gmin, gmax);
  hipFree(gmin);
  hipFree(gmax);
}

__global__ void kernel3(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int chunksize = thread_pixel;
  for (int i = idx*chunksize; i < (idx + 1)*chunksize && i < width*height; i++) {
    int32_t j;
    int counter = 0;
    output[i] = 0;
    for (int k = 0; k < 5; k++) {
      for (j = i - (2-k)*width - 2; j < i - (2-k)*width + 3; j++) {
        if ((j >= 0 && j < width*height) && i/width - j/width + k == 2) {
          output[i] += filter[counter] * input[j];
        }
        counter++;
      }
    }
  }
}

__global__ void normalize3(int32_t *image, int32_t width, int32_t height,
                           int32_t *smallest, int32_t *biggest) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int chunksize = thread_pixel;
  for (int i = idx*chunksize; i < (idx + 1)*chunksize && i < width*height; i++) {
    if (smallest[0] != biggest[0]) {
      image[i] = ((image[i] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
    }
  }
}
