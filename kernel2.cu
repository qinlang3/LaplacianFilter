#include "hip/hip_runtime.h"
#include "kernels.h"
#define threads_block 512


void run_kernel2(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height) {
  // Calculate threads and blocks.
  int num_threads, num_blocks;
  if (width * height > threads_block) {
    num_threads = threads_block;
    num_blocks = (width * height + threads_block - 1) / threads_block;
  }else {
    num_threads = width * height;
    num_blocks = 1;
  }
  dim3 dimBlock(num_threads, 1, 1);
  dim3 dimGrid(num_blocks, 1, 1);
  kernel2<<<dimGrid, dimBlock>>>(filter, dimension, input, output, width, height);
  // Initialiaze golbal device variables to store minimum and maximum.
  int32_t *gmin = NULL;
  int32_t *gmax = NULL;
  hipMalloc((void **)&gmin, width * height * sizeof(int32_t));
  hipMalloc((void **)&gmax, width * height * sizeof(int32_t));
  hipMemcpy(gmin, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);
  hipMemcpy(gmax, output, width * height * sizeof(int32_t), hipMemcpyDeviceToDevice);
  int shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
  reduction<<<dimGrid, dimBlock, shMemSize>>>(gmin, gmax, width * height);

  // Repeat calling reduction kernel to get final maximum and minimum.
  while (num_blocks != 1) {
    int n = num_blocks;
    if (num_blocks > threads_block) {
      num_threads = threads_block;
      num_blocks = (num_blocks + threads_block - 1) / threads_block;
    }else {
      num_threads = num_blocks;
      num_blocks = 1;
    }
    shMemSize = (num_threads <= 32) ? 4 * num_threads * sizeof(int32_t) : 2* num_threads * sizeof(int32_t);
    dim3 newblock(num_threads, 1, 1);
    dim3 newgrid(num_blocks, 1, 1);
    reduction<<<newgrid, newblock, shMemSize>>>(gmin, gmax, n);
  }
  // Calling normalize kernel.
  normalize2<<<dimGrid, dimBlock>>>(output, width, height, gmin, gmax);
  hipFree(gmin);
  hipFree(gmax);
}

__global__ void kernel2(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < width * height) {
    int32_t j;
    int counter = 0;
    output[idx] = 0;
    for (int k = 0; k < 5; k++) {
      for (j = idx - (2-k)*width - 2; j < idx - (2-k)*width + 3; j++) {
        if ((j >= 0 && j < width*height) && ((idx/width)-(j/width)+k == 2)) {
          output[idx] += filter[counter] * input[j]; 
        }
        counter++;
      }
    }
  }
}


__global__ void normalize2(int32_t *image, int32_t width, int32_t height,
                           int32_t *smallest, int32_t *biggest) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < width * height) {
    if (smallest[0] != biggest[0]) {
      image[idx] = ((image[idx] - smallest[0]) * 255) / (biggest[0] - smallest[0]);
    }
  }
}
